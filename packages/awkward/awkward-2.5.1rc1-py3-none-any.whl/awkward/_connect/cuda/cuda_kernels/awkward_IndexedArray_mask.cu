#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

template <typename T, typename C>
__global__ void
awkward_IndexedArray_mask(T* tomask,
                          const C* fromindex,
                          int64_t length,
                          uint64_t invocation_index,
                          uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < length) {
      tomask[thread_id] = fromindex[thread_id] < 0;
    }
  }
}
