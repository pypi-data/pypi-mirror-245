#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

template <typename T, typename C>
__global__ void
awkward_ByteMaskedArray_mask(T* tomask,
                             const C* frommask,
                             int64_t length,
                             bool validwhen,
                             uint64_t invocation_index,
                             uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < length) {
      tomask[thread_id] = (frommask[thread_id] != 0) != validwhen;
    }
  }
}
